#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <stdlib.h>
#include <time.h>


__global__ void monteCarloPiEstimate(int* d_insideCircle, long long int n, unsigned long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    int local_count = 0;

    for (int i = idx; i < n; i += stride) {
        float x = hiprand_uniform(&state) * 2.0f - 1.0f;
        float y = hiprand_uniform(&state) * 2.0f - 1.0f;
        float distance_squared = x * x + y * y;
        if (distance_squared <= 1)
            local_count++;
    }
        atomicAdd(d_insideCircle, local_count);
}


int main() {
    int n = 1 << 24;
    int h_insideCircle = 0;
    int* d_insideCircle;

    hipMalloc(&d_insideCircle, sizeof(int));

    hipMemcpy(d_insideCircle, &h_insideCircle, sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 1024;
    int numBlocks = (n + blockSize - 1) / blockSize;
    monteCarloPiEstimate << <numBlocks, blockSize >> > (d_insideCircle, n, time(NULL));

    hipMemcpy(&h_insideCircle, d_insideCircle, sizeof(int), hipMemcpyDeviceToHost);

    float piEstimate = 4.0f * h_insideCircle / n;
    printf("Estimated Pi = %f\n", piEstimate);

    hipFree(d_insideCircle);

    return 0;
}